#include "hip/hip_runtime.h"
#include "cuda_kernels.h"

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <thrust/device_vector.h>

extern "C"
{
	hipblasHandle_t cublas_handle = nullptr;

	////////////////////////////////////////////////////////////////////////////////////////////////////
	/// <summary> Initialize CUBLAS creating an handle for its context. </summary>
	///
	/// <remarks> Diego Mazala, 15/02/2016. </remarks>
	///
	/// <returns> true if the context has been created, false otherwise. </returns>
	////////////////////////////////////////////////////////////////////////////////////////////////////
	bool cublas_init()
	{
		// Create a handle for CUBLAS
		return (hipblasCreate(&cublas_handle) == HIPBLAS_STATUS_SUCCESS);
	}


	////////////////////////////////////////////////////////////////////////////////////////////////////
	/// <summary> Destroy CUBLAS context. </summary>
	///
	/// <remarks> Diego Mazala, 15/02/2016. </remarks>
	///
	////////////////////////////////////////////////////////////////////////////////////////////////////
	bool cublas_cleanup()
	{
		// Destroy the handle
		return (hipblasDestroy(cublas_handle) == HIPBLAS_STATUS_SUCCESS);
	}



	////////////////////////////////////////////////////////////////////////////////////////////////////
	/// <summary>  Multiply the arrays A and B on GPU and save the result in C </summary>
	///
	/// <remarks> Diego Mazala, 15/02/2016. </remarks>
	///
	/// <param name="A"> Input left matrix. </param>
	/// <param name="B"> Input right matrix. </param>
	/// <param name="C"> Output result matrix. </param>
	///
	////////////////////////////////////////////////////////////////////////////////////////////////////
	bool cublas_matrix_mul(float *dev_C, const float *dev_A, const float *dev_B, const int m, const int k, const int n)
	{
		int lda = m, ldb = k, ldc = m;
		const float alf = 1;
		const float bet = 0;
		const float *alpha = &alf;
		const float *beta = &bet;

		if (!cublas_handle)
			cublas_init();

		// Do the actual multiplication
		return (hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, dev_A, lda, dev_B, ldb, beta, dev_C, ldc) == HIPBLAS_STATUS_SUCCESS);
	}



	//usage example with eigen: matrix_mul(mat_C.data(), mat_A.data(), &vector_of_eigen_vector4[0][0], A.rows(), A.cols(), vector_of_eigen_vector4.size());
	void matrix_mulf(float* mat_c, const float* mat_a, const float* mat_b, int m, int k, int n)
	{
		// transfer to device 
		thrust::device_vector<float> d_a(&mat_a[0], &mat_a[0] + m * k);
		thrust::device_vector<float> d_b(&mat_b[0], &mat_b[0] + k * n);
		thrust::device_vector<float> d_c(&mat_c[0], &mat_c[0] + m * n);

		// Multiply A and B on GPU
		cublas_matrix_mul(thrust::raw_pointer_cast(&d_c[0]), thrust::raw_pointer_cast(&d_a[0]), thrust::raw_pointer_cast(&d_b[0]), m, k, n);

		thrust::copy(d_c.begin(), d_c.end(), &mat_c[0]);
	}

	__global__ void matrixMult(int *a, int *b, int *c, int width)
	{
		int k, sum = 0;
		int col = threadIdx.x + blockDim.x * blockIdx.x;
		int row = threadIdx.y + blockDim.y * blockIdx.y;
		if (col < width && row < width)
		{
			for (k = 0; k < width; k++)
				sum += a[row * width + k] * b[k * width + col];
			c[row * width + col] = sum;
		}
	}

	__global__ void compute_pixel_depth_kernel(
		float* in_out_depth_buffer_1f, 
		const float* in_world_points_4f, 
		const float* in_clip_points_4f, 
		unsigned int point_count, 
		unsigned int window_width, 
		unsigned int window_height)
	{
		// unique block index inside a 3D block grid
		//const unsigned long long int blockId = blockIdx.x //1D
		//	+ blockIdx.y * gridDim.x //2D
		//	+ gridDim.x * gridDim.y * blockIdx.z; //3D
		//// global unique thread index, block dimension uses only x-coordinate
		//const unsigned long long int threadId = blockId * blockDim.x + threadIdx.x;
		
		const unsigned long long int threadId = blockIdx.x * blockDim.x + threadIdx.x;

		const unsigned int pixel_count = window_width * window_height;

		if (threadId >= point_count)
			return;
		
		const float clip_x = in_clip_points_4f[threadId * 4 + 0];	
		const float clip_y = in_clip_points_4f[threadId * 4 + 1];
		const float clip_z = in_clip_points_4f[threadId * 4 + 2];
		const float clip_w = in_clip_points_4f[threadId * 4 + 3];
		const float ndc_x = clip_x / clip_w;
		const float ndc_y = clip_y / clip_w;
		const float ndc_z = clip_z / clip_w;

		if (ndc_x < -1 || ndc_x > 1 || ndc_y < -1 || ndc_y > 1 || ndc_z < -1 || ndc_z > 1)
			return;

		const float pixel_x = window_width / 2.0f * ndc_x + window_width / 2.0f;
		const float pixel_y = window_height / 2.0f * ndc_y + window_height / 2.0f;

		const int depth_index = (int)pixel_y * window_width + (int)pixel_x;

		if (depth_index > 0 && depth_index < pixel_count)
		{
			const float& curr_depth = fabs(in_out_depth_buffer_1f[depth_index]);
			const float& new_depth = fabs(in_world_points_4f[threadId * 4 + 2]);	// z coord
			__syncthreads();

			if (new_depth < curr_depth)
			{
				in_out_depth_buffer_1f[depth_index] = new_depth;
			}
		}
	}


	void compute_depth_buffer(	
			float* depth_buffer, 
			const float* world_points_4f, 
			unsigned int point_count, 
			const float* projection_mat4x4, 
			unsigned int window_width, 
			unsigned int window_height)
	{
		const unsigned int pixel_count = window_width * window_height;

		// transfer to device 
		thrust::device_vector<float> d_depth_buffer(&depth_buffer[0], &depth_buffer[0] + pixel_count);
		thrust::device_vector<float> d_projection_mat(&projection_mat4x4[0], &projection_mat4x4[0] + 16);
		
		thrust::device_vector<float> d_world_points(&world_points_4f[0], &world_points_4f[0] + point_count * 4);
		thrust::device_vector<float> d_clip_points(&world_points_4f[0], &world_points_4f[0] + point_count * 4);
		
		cublas_matrix_mul(thrust::raw_pointer_cast(&d_clip_points[0]), thrust::raw_pointer_cast(&d_projection_mat[0]), thrust::raw_pointer_cast(&d_world_points[0]), 4, 4, point_count);

		unsigned int threads_per_block = 1024;
		unsigned int num_blocks = 1 + point_count / threads_per_block;

		compute_pixel_depth_kernel <<< num_blocks, threads_per_block >>> (
			thrust::raw_pointer_cast(&d_depth_buffer[0]),
			thrust::raw_pointer_cast(&d_world_points[0]),
			thrust::raw_pointer_cast(&d_clip_points[0]), 
			point_count,
			window_width, 
			window_height);

		thrust::copy(d_depth_buffer.begin(), d_depth_buffer.end(), &depth_buffer[0]);
	}



	void make_identity_4x4(float *mat)
	{
		for (int i = 0; i < 16; ++i)
			mat[i] = 0;


		mat[0] = mat[5] = mat[10] = mat[15] = 1.0f;
	}



	//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
	void print_matrix(const float *A, int nr_rows_A, int nr_cols_A)
	{
		for (int i = 0; i < nr_rows_A; ++i)
		{
			for (int j = 0; j < nr_cols_A; ++j)
			{
				std::cout << A[j * nr_rows_A + i] << " ";
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}

};