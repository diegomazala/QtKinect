#ifndef _KINECT_CUDA_KERNELS_CU_
#define _KINECT_CUDA_KERNELS_CU_

#include "KinectFusionKernels.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>


ushort* depth_buffer = 0;
size_t depth_pitch;
ushort depth_width;
ushort depth_height;


extern "C"
{
	void knt_cuda_allocate()
	{
		// allocate memory in gpu for depth buffer
		checkCudaErrors(
			hipMallocPitch(
			&depth_buffer,
			&depth_pitch,
			sizeof(ushort) * depth_width,
			depth_height));



	}


	void knt_cuda_free()
	{
		checkCudaErrors(hipFree(depth_buffer));
	}
}

#endif // #ifndef _KINECT_CUDA_KERNELS_CU_
